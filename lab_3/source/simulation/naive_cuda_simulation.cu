#include "hip/hip_runtime.h"
#include "naive_cuda_simulation.cuh"
#include "physics/gravitation.h"
#include "physics/mechanics.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_wrappers.cuh"
#include "simulation/constants.h"

#include <exception>

void NaiveCudaSimulation::allocate_device_memory(Universe &universe, void **d_weights, void **d_forces, void **d_velocities, void **d_positions)
{
    auto error = parprog_cudaMalloc(d_weights, universe.num_bodies * sizeof(double));
    error = parprog_cudaMalloc(d_forces, universe.num_bodies * sizeof(double2));
    error = parprog_cudaMalloc(d_velocities, universe.num_bodies * sizeof(double2));
    error = parprog_cudaMalloc(d_positions, universe.num_bodies * sizeof(double2));
}

void NaiveCudaSimulation::free_device_memory(void **d_weights, void **d_forces, void **d_velocities, void **d_positions)
{
    auto error = parprog_cudaFree(*d_weights);
    d_weights = nullptr;
    error = parprog_cudaFree(*d_forces);
    d_forces = nullptr;
    error = parprog_cudaFree(*d_velocities);
    d_velocities = nullptr;
    error = parprog_cudaFree(*d_positions);
    d_positions = nullptr;
}

void NaiveCudaSimulation::copy_data_to_device(Universe &universe, void *d_weights, void *d_forces, void *d_velocities, void *d_positions)
{
    auto error = parprog_cudaMemcpy(d_weights, universe.weights.data(), universe.num_bodies * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }
    // convert the Vector2d<double> to double2
    std::vector<double2> converted{universe.num_bodies};
    for (int i = 0; i < universe.num_bodies; ++i)
    {
        converted[i] = make_double2(universe.forces[i][0], universe.forces[i][1]);
    }

    error = parprog_cudaMemcpy(d_forces, converted.data(), universe.num_bodies * sizeof(double2), hipMemcpyKind::hipMemcpyHostToDevice);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }
    for (int i = 0; i < universe.num_bodies; ++i)
    {
        converted[i] = make_double2(universe.velocities[i][0], universe.velocities[i][1]);
    }
    error = parprog_cudaMemcpy(d_velocities, converted.data(), universe.num_bodies * sizeof(double2), hipMemcpyKind::hipMemcpyHostToDevice);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }
    for (int i = 0; i < universe.num_bodies; ++i)
    {
        converted[i] = make_double2(universe.positions[i][0], universe.positions[i][1]);
    }
    error = parprog_cudaMemcpy(d_positions, converted.data(), universe.num_bodies * sizeof(double2), hipMemcpyKind::hipMemcpyHostToDevice);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }
}

void NaiveCudaSimulation::copy_data_from_device(Universe &universe, void *d_weights, void *d_forces, void *d_velocities, void *d_positions)
{
    auto error = parprog_cudaMemcpy(universe.weights.data(), d_weights, universe.num_bodies * sizeof(double), hipMemcpyKind::hipMemcpyDeviceToHost);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }

    std::vector<double2> converted{universe.num_bodies};

    error = parprog_cudaMemcpy(converted.data(), d_forces, universe.num_bodies * sizeof(double2), hipMemcpyKind::hipMemcpyDeviceToHost);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }
    for (int i = 0; i < universe.num_bodies; ++i)
    {
        universe.forces[i].set(converted[i].x, converted[i].y);
    }

    error = parprog_cudaMemcpy(converted.data(), d_velocities, universe.num_bodies * sizeof(double2), hipMemcpyKind::hipMemcpyDeviceToHost);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }
    for (int i = 0; i < universe.num_bodies; ++i)
    {
        universe.velocities[i].set(converted[i].x, converted[i].y);
    }

    error = parprog_cudaMemcpy(converted.data(), d_positions, universe.num_bodies * sizeof(double2), hipMemcpyKind::hipMemcpyDeviceToHost);
    if (error != 0)
    {
        throw std::runtime_error(hipGetErrorString(error));
    }
    for (int i = 0; i < universe.num_bodies; ++i)
    {
        universe.positions[i].set(converted[i].x, converted[i].y);
    }
}

__global__ void calculate_forces_kernel(std::uint32_t num_bodies, double2 *d_positions, double *d_weights, double2 *d_forces)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_bodies)
        return;
    double2 body_position = d_positions[i];
    // get body mass
    double body_mass = d_weights[i];

    double2 applied_force_vector = make_double2(0, 0);

    for (int distant_body_idx = 0; distant_body_idx < num_bodies; distant_body_idx++)
    {
        if (i == distant_body_idx)
        {
            continue;
        }
        // get distant body positions
        double2 distant_body_position = d_positions[distant_body_idx];

        // calculate vector between bodies to get the direction of the gravitational force
        double2 direction_vector = make_double2(distant_body_position.x - body_position.x, distant_body_position.y - body_position.y);

        // calculate the distance between the bodies
        double distance = sqrt(pow(direction_vector.x, 2) + pow(direction_vector.y, 2));

        // calculate gravitational force between the bodies
        double force = gravitational_constant * (body_mass * d_weights[distant_body_idx]) / pow(distance, 2);

        // create the force vector
        double2 force_vector = make_double2(direction_vector.x * (force / distance), direction_vector.y * (force / distance));

        // sum forces applied to body
        applied_force_vector = make_double2(applied_force_vector.x + force_vector.x, applied_force_vector.y + force_vector.y);
    }

    // store applied force
    d_forces[i] = applied_force_vector;
}

void NaiveCudaSimulation::calculate_forces(Universe &universe, void *d_positions, void *d_weights, void *d_forces)
{
    std::uint32_t block_dim = 512;
    std::uint32_t grid_dim;

    if (universe.num_bodies % block_dim == 0)
    {
        grid_dim = universe.num_bodies / block_dim;
    }
    else
    {
        grid_dim = (universe.num_bodies - (universe.num_bodies % block_dim) + block_dim) / block_dim;
    }
    calculate_forces_kernel<<<grid_dim, block_dim>>>(universe.num_bodies, (double2 *)d_positions, (double *)d_weights, (double2 *)d_forces);
    hipDeviceSynchronize();
}

__global__ void calculate_velocities_kernel(std::uint32_t num_bodies, double2 *d_forces, double *d_weights, double2 *d_velocities)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_bodies)
        return;
    d_velocities[i].x = d_velocities[i].x + d_forces[i].x / d_weights[i] * epoch_in_seconds;
    d_velocities[i].y = d_velocities[i].y + d_forces[i].y / d_weights[i] * epoch_in_seconds;
}

void NaiveCudaSimulation::calculate_velocities(Universe &universe, void *d_forces, void *d_weights, void *d_velocities)
{
    std::uint32_t block_dim = 512;
    std::uint32_t grid_dim;

    if (universe.num_bodies % block_dim == 0)
    {
        grid_dim = universe.num_bodies / block_dim;
    }
    else
    {
        grid_dim = (universe.num_bodies - (universe.num_bodies % block_dim) + block_dim) / block_dim;
    }
    calculate_velocities_kernel<<<grid_dim, block_dim>>>(universe.num_bodies, (double2 *)d_forces, (double *)d_weights, (double2 *)d_velocities);
    hipDeviceSynchronize();
}

__global__ void calculate_positions_kernel(std::uint32_t num_bodies, double2 *d_velocities, double2 *d_positions)
{
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_bodies)
        return;

    // update position
    d_positions[i].x = d_positions[i].x + d_velocities[i].x * epoch_in_seconds;
    d_positions[i].y = d_positions[i].y + d_velocities[i].y * epoch_in_seconds;
}

void NaiveCudaSimulation::calculate_positions(Universe &universe, void *d_velocities, void *d_positions)
{
    std::uint32_t block_dim = 512;
    std::uint32_t grid_dim;

    if (universe.num_bodies % block_dim == 0)
    {
        grid_dim = universe.num_bodies / block_dim;
    }
    else
    {
        grid_dim = (universe.num_bodies - (universe.num_bodies % block_dim) + block_dim) / block_dim;
    }
    calculate_positions_kernel<<<grid_dim, block_dim>>>(universe.num_bodies, (double2 *)d_velocities, (double2 *)d_positions);
    hipDeviceSynchronize();
}

void NaiveCudaSimulation::simulate_epochs(Plotter &plotter, Universe &universe, std::uint32_t num_epochs, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs)
{
    void *d_weights;
    void *d_forces;
    void *d_velocities;
    void *d_positions;

    allocate_device_memory(universe, &d_weights, &d_forces, &d_velocities, &d_positions);
    copy_data_to_device(universe, d_weights, d_forces, d_velocities, d_positions);
    for (int i = 0; i < num_epochs; i++)
    {
        simulate_epoch(plotter, universe, create_intermediate_plots, plot_intermediate_epochs, d_weights, d_forces, d_velocities, d_positions);
    }
    copy_data_from_device(universe, d_weights, d_forces, d_velocities, d_positions);
    free_device_memory(&d_weights, &d_forces, &d_velocities, &d_positions);
}

__global__ void get_pixels_kernel(std::uint32_t num_bodies, double2 *d_positions, std::uint8_t *d_pixels, std::uint32_t plot_width, std::uint32_t plot_height, double plot_bounding_box_x_min, double plot_bounding_box_x_max, double plot_bounding_box_y_min, double plot_bounding_box_y_max)
{
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_bodies)
        return;
    auto pos = d_positions[i];
    if (pos.x > plot_bounding_box_x_max || pos.x < plot_bounding_box_x_min || pos.y > plot_bounding_box_y_max || pos.y < plot_bounding_box_y_min)
    {
        return;
    }
    std::size_t pixel_x = ((pos.x - plot_bounding_box_x_min) / (plot_bounding_box_x_max - plot_bounding_box_x_min)) * (plot_width - 1);
    std::size_t pixel_y = ((pos.y - plot_bounding_box_y_min) / (plot_bounding_box_y_max - plot_bounding_box_y_min)) * (plot_height - 1);
    d_pixels[pixel_x + pixel_y * plot_width] = 1;
}

std::vector<std::uint8_t> NaiveCudaSimulation::get_pixels(std::uint32_t plot_width, std::uint32_t plot_height, BoundingBox plot_bounding_box, void *d_positions, std::uint32_t num_bodies)
{
    auto plot_size = plot_width * plot_height;
    void *d_pixels;
    parprog_cudaMalloc(&d_pixels, plot_size * sizeof(std::uint8_t));
    std::vector<std::uint8_t> pixels(plot_size, 0);
    parprog_cudaMemcpy(d_pixels, pixels.data(), plot_size * sizeof(std::uint8_t), hipMemcpyKind::hipMemcpyHostToDevice);
    std::uint32_t block_dim = 512;
    std::uint32_t grid_dim;

    if (num_bodies % block_dim == 0)
    {
        grid_dim = num_bodies / block_dim;
    }
    else
    {
        grid_dim = (num_bodies - (num_bodies % block_dim) + block_dim) / block_dim;
    }
    get_pixels_kernel<<<grid_dim, block_dim>>>(num_bodies, (double2 *)d_positions, (std::uint8_t *)d_pixels, plot_width, plot_height, plot_bounding_box.x_min, plot_bounding_box.x_max, plot_bounding_box.y_min, plot_bounding_box.y_max);
    hipDeviceSynchronize();
    parprog_cudaMemcpy(pixels.data(), d_pixels, plot_size * sizeof(std::uint8_t), hipMemcpyKind::hipMemcpyDeviceToHost);
    parprog_cudaFree(d_pixels);
    return pixels;
}
__global__ void compress_pixels_kernel(std::uint32_t num_raw_pixels, std::uint8_t *d_raw_pixels, std::uint8_t *d_compressed_pixels)
{
    std::uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ std::uint8_t pixels[8];
    pixels[threadIdx.x] = 0;
    __syncthreads();
    if (d_raw_pixels[i] != 0)
    {
        pixels[threadIdx.x] = pixels[threadIdx.x] | (1 << threadIdx.x);
    }
    __syncthreads();
    if(threadIdx.x == 0){
        std::uint8_t pixel = 0;
        for(int i = 0; i < blockDim.x; ++i){
            pixel |= pixels[i];
        }
        d_compressed_pixels[blockIdx.x] = pixel;
    }
}

void NaiveCudaSimulation::compress_pixels(std::vector<std::uint8_t> &raw_pixels, std::vector<std::uint8_t> &compressed_pixels)
{
    auto num_raw_pixels = raw_pixels.size();
    void *d_raw_pixels;
    void *d_compressed_pixels;
    parprog_cudaMalloc(&d_raw_pixels, num_raw_pixels);
    parprog_cudaMalloc(&d_compressed_pixels, num_raw_pixels / 8);
    parprog_cudaMemcpy(d_raw_pixels, raw_pixels.data(), num_raw_pixels, hipMemcpyKind::hipMemcpyHostToDevice);
    parprog_cudaMemcpy(d_compressed_pixels, compressed_pixels.data(), num_raw_pixels / 8, hipMemcpyKind::hipMemcpyHostToDevice);
    std::uint32_t block_dim = 8;
    std::uint32_t grid_dim= num_raw_pixels / block_dim;
    compress_pixels_kernel<<<grid_dim, block_dim>>>(num_raw_pixels, (std::uint8_t *)d_raw_pixels, (std::uint8_t *)d_compressed_pixels);
    hipDeviceSynchronize();

    parprog_cudaMemcpy(compressed_pixels.data(), d_compressed_pixels, num_raw_pixels / 8, hipMemcpyKind::hipMemcpyDeviceToHost);

    parprog_cudaFree(d_compressed_pixels);
    parprog_cudaFree(d_raw_pixels);
}

void NaiveCudaSimulation::simulate_epoch(Plotter &plotter, Universe &universe, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs, void *d_weights, void *d_forces, void *d_velocities, void *d_positions)
{
    calculate_forces(universe, d_positions, d_weights, d_forces);
    calculate_velocities(universe, d_forces, d_weights, d_velocities);
    calculate_positions(universe, d_velocities, d_positions);

    universe.current_simulation_epoch++;
    if (create_intermediate_plots)
    {
        if (universe.current_simulation_epoch % plot_intermediate_epochs == 0)
        {
            std::vector<std::uint8_t> pixels = get_pixels(plotter.get_plot_width(), plotter.get_plot_height(), plotter.get_plot_bounding_box(), d_positions, universe.num_bodies);
            plotter.add_active_pixels_to_image(pixels);

            // This is a dummy to use compression in plotting, although not beneficial performance-wise
            // ----
            // std::vector<std::uint8_t> compressed_pixels;
            // compressed_pixels.resize(pixels.size()/8);
            // compress_pixels(pixels, compressed_pixels);
            // plotter.add_compressed_pixels_to_image(compressed_pixels);
            // ----

            plotter.write_and_clear();
        }
    }
}

void NaiveCudaSimulation::calculate_forces_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void *d_positions, void *d_weights, void *d_forces)
{
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_forces_kernel<<<gridDim, blockDim>>>(num_bodies, (double2 *)d_positions, (double *)d_weights, (double2 *)d_forces);
}

void NaiveCudaSimulation::calculate_velocities_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void *d_forces, void *d_weights, void *d_velocities)
{
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_velocities_kernel<<<gridDim, blockDim>>>(num_bodies, (double2 *)d_forces, (double *)d_weights, (double2 *)d_velocities);
}

void NaiveCudaSimulation::calculate_positions_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void *d_velocities, void *d_positions)
{
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_positions_kernel<<<gridDim, blockDim>>>(num_bodies, (double2 *)d_velocities, (double2 *)d_positions);
}
